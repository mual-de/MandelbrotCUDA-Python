#include "hip/hip_runtime.h"
#include "../include/MandelbrotCUDA.hpp"
#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/system/system_error.h>
#include <thrust/iterator/counting_iterator.h>

using namespace MandelbrotCUDA;
std::vector<std::vector<unsigned char>> colorizeMandelbrotThrustDev(const thrust::device_vector<int> devValues, const int maxIter);

std::vector<int> MandelbrotCUDA::createMandelbrotThrust(const MandelbrotConfig config){
    return std::vector<int>(1);
}


__global__
void createMandelbrotCUDADevice(const int width, const int height, const int maxIter, const size_t pitch, const double maxSqr, const double* xVals, const double* yVals, int* results){
    int posX = blockIdx.x*blockDim.x + threadIdx.x;
    int posY = blockIdx.y*blockDim.y + threadIdx.y;
    if(posX >= width || posY >= height){
        return;
    }
    const double re = xVals[posX];
    const double im = yVals[posY];
    int remainIter = maxIter;
    double reSqr = re * re;
    double imSqr = im * im;
    double reIm = re * im;
    double max2 = reSqr + imSqr;
    while ((max2 <= maxSqr) && (remainIter > 0))
    {
        remainIter--;
        double xBetween = reSqr - imSqr + re;
        double yBetween = reIm + reIm + im;
        reSqr = xBetween * xBetween;
        imSqr = yBetween * yBetween;
        reIm = xBetween * yBetween;
        max2 = reSqr + imSqr;
    }
    results[posX+posY*pitch] = maxIter - remainIter;

}

void calculatePositionVectors(thrust::device_vector<double>& xVals, thrust::device_vector<double>& yVals, const MandelbrotConfig config){
    double reRef = (config.reMax - config.reMin) / static_cast<double>(config.width);
    double imRef = (config.imMax - config.imMin) / static_cast<double>(config.height);
    thrust::device_vector<double> tempX(xVals.size());
    thrust::counting_iterator<int> startX(0);
    thrust::counting_iterator<int> endX = startX + xVals.size();
    thrust::device_vector<double> tempY(yVals.size());
    thrust::counting_iterator<int> startY(0);
    thrust::counting_iterator<int> endY = startY + yVals.size();
    // Precalculate positions in image regarding re/im values
    thrust::fill(xVals.begin(), xVals.end(), config.reMin);
    thrust::fill(tempX.begin(), tempX.end(), reRef);
    thrust::transform(startX, endX, tempX.begin(), tempX.begin(), thrust::multiplies<double>());
    thrust::transform(tempX.begin(), tempX.end(), xVals.begin(), xVals.begin(), thrust::plus<double>());
    thrust::fill(yVals.begin(), yVals.end(), config.imMin);
    thrust::fill(tempY.begin(), tempY.end(), imRef);
    thrust::transform(startY, endY, tempY.begin(), tempY.begin(), thrust::multiplies<double>());
    thrust::transform(tempY.begin(), tempY.end(), yVals.begin(), yVals.begin(), thrust::plus<double>());
}

std::vector<int> MandelbrotCUDA::createMandelbrotCUDAPitch(const MandelbrotConfig config){
    int* calcMem = 0;
    thrust::device_vector<double> xVals(config.width);
    thrust::device_vector<double> yVals(config.height);
    size_t memPitch;
    hipError_t err;
    err = hipMallocPitch((void **)&calcMem, &memPitch, static_cast<size_t>(config.width * sizeof(int)), static_cast<size_t>(config.height));
    if(err != hipSuccess){
        return std::vector<int>(1);
    }
    
    calculatePositionVectors(xVals, yVals, config);
    
    double*   xValsPtr = thrust::raw_pointer_cast(xVals.data());
    double*  yValsPtr = thrust::raw_pointer_cast(yVals.data());
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks(config.width / threadsPerBlock.x, config.height / threadsPerBlock.y);
    createMandelbrotCUDADevice<<<numBlocks, threadsPerBlock>>>(config.width, config.height, config.maxIter,memPitch/sizeof(int), config.maxSqr, xValsPtr, yValsPtr, calcMem);
    err = hipDeviceSynchronize();

    std::vector<int> ret(config.height*config.width);
    err = hipMemcpy2D(ret.data(), config.width*sizeof(int), calcMem, memPitch, config.width * sizeof(int), config.height, hipMemcpyDeviceToHost);
    hipFree(calcMem);
    return ret;
}

std::vector<int> MandelbrotCUDA::createMandelbrotCUDA(const MandelbrotConfig config){
    int* calcMem = 0;
    thrust::device_vector<double> xVals(config.width);
    thrust::device_vector<double> yVals(config.height);
    size_t memPitch = config.width*sizeof(int);
    hipError_t err;
    err = hipMalloc((void **)&calcMem, static_cast<size_t>(config.width * sizeof(int)*config.height));
    if(err != hipSuccess){
        return std::vector<int>(1);
    }
    
    calculatePositionVectors(xVals, yVals, config);
    
    double*   xValsPtr = thrust::raw_pointer_cast(xVals.data());
    double*  yValsPtr = thrust::raw_pointer_cast(yVals.data());
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks(config.width / threadsPerBlock.x, config.height / threadsPerBlock.y);
    createMandelbrotCUDADevice<<<numBlocks, threadsPerBlock>>>(config.width, config.height, config.maxIter,config.width, config.maxSqr, xValsPtr, yValsPtr, calcMem);
    err = hipDeviceSynchronize();

    std::vector<int> ret(config.height*config.width);
    err = hipMemcpy2D(ret.data(), config.width*sizeof(int), calcMem, memPitch, config.width * sizeof(int), config.height, hipMemcpyDeviceToHost);
    hipFree(calcMem);
    return ret;
}


std::vector<std::vector<unsigned char>> MandelbrotCUDA::createMandelbrotCUDAColorized(const MandelbrotConfig config){
    thrust::device_vector<int> calcMem(config.width * config.height);
    thrust::device_vector<double> xVals(config.width);
    thrust::device_vector<double> yVals(config.height);
    size_t memPitch = config.width*sizeof(int);
    hipError_t err;

    
    calculatePositionVectors(xVals, yVals, config);

    double*   xValsPtr = thrust::raw_pointer_cast(xVals.data());
    double*  yValsPtr = thrust::raw_pointer_cast(yVals.data());
    int* calcMemPtr = thrust::raw_pointer_cast(calcMem.data());
    
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks(config.width / threadsPerBlock.x, config.height / threadsPerBlock.y);
    createMandelbrotCUDADevice<<<numBlocks, threadsPerBlock>>>(config.width, config.height, config.maxIter,config.width, config.maxSqr, xValsPtr, yValsPtr, calcMemPtr);
    err = hipDeviceSynchronize();
    return colorizeMandelbrotThrustDev(calcMem, config.maxIter);
}


/**
 * @brief set red color amount
 * 
 * Colors from: https://stackoverflow.com/questions/16500656/which-color-gradient-is-used-to-color-mandelbrot-in-wikipedia
 * Special thanks to q9f for reverse engineering!
 */
struct color_red : public thrust::unary_function<int, unsigned char>
{
    const unsigned char colorVals[16] = {   66,25,9,4,0,12,24,57,134,211,241,248,255,204,153,106};
    const int max;
    color_red(int _max) : max(_max) {}

    __host__ __device__
        unsigned char operator()(const int& x) const {
            if(x == 0 || x == max){
                return 0;
            }
            int i = x % 16;
            return colorVals[i];
        }
};

/**
 * @brief set green color amount
 * 
 * Colors from: https://stackoverflow.com/questions/16500656/which-color-gradient-is-used-to-color-mandelbrot-in-wikipedia
 * Special thanks to q9f for reverse engineering!
 */
struct color_green : public thrust::unary_function<int, unsigned char>
{
    const unsigned char colorVals[16] = { 30,7,1,4,7,44,82,125,181,236,233,201,170,128,87,52 };
    const int max;
    color_green(int _max) : max(_max) {}

    __host__ __device__
        unsigned char operator()(const int& x) const {
            if(x == 0 || x == max){
                return 0;
            }
            int i = x % 16;
            return colorVals[i];
        }
};

/**
 * @brief set blue color amount
 * 
 * Colors from: https://stackoverflow.com/questions/16500656/which-color-gradient-is-used-to-color-mandelbrot-in-wikipedia
 * Special thanks to q9f for reverse engineering!
 */
struct color_blue : public thrust::unary_function<int, unsigned char>
{
    const unsigned char colorVals[16] = { 15,26,47,73,100,138,177,209,229,248,191,95,0,0,0,3 };
    const int max;
    color_blue(int _max) : max(_max) {}

    __host__ __device__
        unsigned char operator()(const int& x) const {
            if(x == 0 || x == max){
                return 0;
            }
            int i = x % 16;
            return colorVals[i];
        }
};

/**
 * @brief Colorize all pixels based on the red, green and blue color function.
 * 
 * @param devValues device_vector containing result from cuda based calculation.
 * @param maxIter maximum iteration count
 * @return std::vector<std::vector<unsigned char>> r,g,b values
 */
std::vector<std::vector<unsigned char>> colorizeMandelbrotThrustDev(const thrust::device_vector<int> devValues, const int maxIter){
    thrust::device_vector<unsigned char> devRed(devValues.size());
    thrust::device_vector<unsigned char> devGreen(devValues.size());
    thrust::device_vector<unsigned char> devBlue(devValues.size());
    thrust::transform(devValues.begin(), devValues.end(), devRed.begin(), color_red(maxIter));
    thrust::transform(devValues.begin(), devValues.end(), devGreen.begin(), color_green(maxIter));
    thrust::transform(devValues.begin(), devValues.end(), devBlue.begin(), color_blue(maxIter));
    std::vector<std::vector<unsigned char>> result(3, std::vector<unsigned char>(devValues.size()));
    thrust::copy(devRed.begin(), devRed.end(), result[0].begin());
    thrust::copy(devGreen.begin(), devGreen.end(), result[1].begin());
    thrust::copy(devBlue.begin(), devBlue.end(), result[2].begin());
    return result;
}

/**
 * @brief Colorize mandelbrot results.
 * 
 * @param values mandelbrot calculation results as std::vector
 * @param maxIter maximum iteration
 * @return std::vector<std::vector<unsigned char>> r,g,b color vectors.
 */
std::vector<std::vector<unsigned char>> MandelbrotCUDA::colorizeMandelbrotThrust(const std::vector<int> values, const int maxIter){
    thrust::device_vector<int> devValues(values.begin(), values.end());
    thrust::device_vector<unsigned char> devRed(values.size());
    thrust::device_vector<unsigned char> devGreen(values.size());
    thrust::device_vector<unsigned char> devBlue(values.size());
    thrust::transform(devValues.begin(), devValues.end(), devRed.begin(), color_red(maxIter));
    thrust::transform(devValues.begin(), devValues.end(), devGreen.begin(), color_green(maxIter));
    thrust::transform(devValues.begin(), devValues.end(), devBlue.begin(), color_blue(maxIter));
    //thrust::copy(devRes.begin(), devRes.end(), hstRes.begin());
    //thrust::copy(hstRes.begin(), hstRes.end(), result.begin());
    // why error? thrust::copy(devValues.begin(), devValues.end(), result.begin());
    std::vector<std::vector<unsigned char>> result(3, std::vector<unsigned char>(values.size()));
    thrust::copy(devRed.begin(), devRed.end(), result[0].begin());
    thrust::copy(devGreen.begin(), devGreen.end(), result[1].begin());
    thrust::copy(devBlue.begin(), devBlue.end(), result[2].begin());
    return result;
}